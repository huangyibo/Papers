#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N)
{
#define STRIDE       32
#define OFFSET        0
#define GROUP_SIZE  512
  int n_elem_per_thread = N / (gridDim.x * blockDim.x);
  int block_start_idx = n_elem_per_thread * blockIdx.x * blockDim.x;
  int thread_start_idx = block_start_idx
            + (threadIdx.x / STRIDE) * n_elem_per_thread * STRIDE
            + ((threadIdx.x + OFFSET) % STRIDE);
  int thread_end_idx = thread_start_idx + n_elem_per_thread * STRIDE;
  if(thread_end_idx > N) thread_end_idx = N;
  int group = (threadIdx.x / GROUP_SIZE) & 1;
  for(int idx=thread_start_idx; idx < thread_end_idx; idx+=STRIDE)
  {
    if(!group) a[idx] = a[idx] * a[idx];
    else       a[idx] = a[idx] + a[idx];
  }
}
// main routine that executes on the host
int main(void)
{
  float *a_h, *a_d;  // Pointer to host & device arrays
  const int N = 1<<25;  // Make a big array with 2**N elements
  size_t size = N * sizeof(float);
  a_h = (float *)malloc(size);        // Allocate array on host
  hipMalloc((void **) &a_d, size);   // Allocate array on device
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++) a_h[i] = (float)i;
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  
  // Set number of threads and blocks
  int n_threads_per_block = 1<<9;  // 512 threads per block
  int n_blocks = 1<<10;  // 1024 blocks
  // Do calculation on device
  square_array <<< n_blocks, n_threads_per_block >>> (a_d, N);
  hipDeviceSynchronize();  // Wait for square_array to finish on CUDA
  // Retrieve result from device and store it in host array
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
  // Print some of the results and the CUDA execution time
  for (int i=0; i<N; i+=N/50) printf("%d %f\n", i, a_h[i]);
  // Cleanup
  free(a_h); hipFree(a_d);
}

