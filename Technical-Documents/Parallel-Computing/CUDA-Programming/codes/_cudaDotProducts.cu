//nvcc -o test -I/usr/local/cuda/include -L/usr/local/cuda/lib64 _cudaDotProducts.cu -lcuda -lcublas
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

int main(void)
{
    const int nvals = 10;
    const size_t sz = sizeof(double) * (size_t)nvals;
    double x[nvals], y[nvals];
    double *x_, *y_, *result_;
    double result=0., resulth=0.;

    for(int i=0; i<nvals; i++) {
        x[i] = y[i] = (double)(i)/(double)(nvals);
        resulth += x[i] * y[i];
    }

    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);

    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);
    hipMalloc( (void **)(&result_), sizeof(double) );

    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    hipblasDdot(h, nvals, x_, 1, y_, 1, result_);

    hipMemcpy(&result, result_, sizeof(double), hipMemcpyDeviceToHost);

    printf("%f %f\n", resulth, result);

    return 0;
}
