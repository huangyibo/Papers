#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DATA_SIZE 1048576     //该数值随便指定
#define THREAD_NUM 256
int data[DATA_SIZE];    //全局data数组，要计算数组中数据的平方和

/*
  * @author Junpeng Zhu
  * @功能  为当前程序设置一个合适的GPU，当然还有其它的方式能够实现该功能，比如设置环境变量
*/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);   //统计计算机中支持CUDA的显卡的个数
  if (count == 0){
    fprintf(stderr, "There is no device!\n");
    return false;
  }
  
  int i;
  for (i = 0; i < count; i++){
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop,i) == hipSuccess){
      if (prop.major >= 1 and prop.minor >= 0){
        break;
      }
    }
  }
  if (i == count){
    fprintf(stderr,"There is no device support CUDA 1.x.\n");
    return false;
  }

  hipSetDevice(i);   //设定满足>=1.0的计算设备为当前程序需要的计算设备
  return true;
}

/*
  * @author Junpeng Zhu
  * @功能  随机数生成函数，生成0-9的以内的整数
*/
void GenerateNumbers(int *number, int size){
  srand((unsigned)time(NULL));
  for (int i =0; i < size; i++){
    number[i] = rand() % 10;
  }
}

/*
 * kernel function is performced in the GPU device.
 * @author Junpeng Zhu
*/
__global__ static void sumOfSquares(int* num, int* result, clock_t* time){
  const int tid = threadIdx.x;    //threadIdx是CUDA的内建变量，表示当前的thread是第几个thread（从0开始计算）
  const int size = DATA_SIZE / THREAD_NUM;
  int sum = 0;
  int i;
  clock_t start;
  if (tid == 0) start = clock();
  for (i = tid*size; i < (tid+1)*size; i++){
    sum += num[i] * num[i];
  }

  result[tid] = sum;
  if (tid == 0) *time = clock()-start;
}


int main(){
  if (!InitCUDA()){
    return 0;
  }else{
    printf("CUDA initialized.\n");
    GenerateNumbers(data, DATA_SIZE);   //生成指定规模的随机数据，所有数据范围均在0-9
    int *gpudata, *result;
    clock_t* time;
    if (!(hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE) == hipSuccess)){
      fprintf(stderr, "Memory error.\n");
      return 0;
    }//在device上分配一块linear地址空间，地址为&gpudata，每个单元大小为int，总共DATA_SIZE个单元
    if(!(hipMalloc((void **) &result, sizeof(int)* THREAD_NUM)==hipSuccess)){
      fprintf(stderr, "Memory error.\n");
      return 0;
    }  //在device上分配内存，大小为int
    if(!(hipMalloc((void**) &time, sizeof(clock_t)) == hipSuccess)){
      fprintf(stderr, "Memory error.\n");
      return 0;
    }
    if(!(hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice)== hipSuccess)){
      fprintf(stderr, "Memory error.\n");
      return 0;
    }  //从host的data中将DATA_SIZE个单元的数据复制到device的gpudata中，这是输入数据，需要从host中获取
    
    sumOfSquares<<<1,THREAD_NUM,0>>>(gpudata,result,time);
    int sum_gpu[THREAD_NUM];  // 将GPU计算结果拷贝到该变量中
    clock_t used_time;
    hipMemcpy(&sum_gpu, result, sizeof(int)*THREAD_NUM, hipMemcpyDeviceToHost);   //将GPU中的result值拷贝到CPU的sum变量中
    hipMemcpy(&used_time, time, sizeof(clock_t), hipMemcpyDeviceToHost);
    
    int final_sum = 0;
    for (int i = 0; i < THREAD_NUM; i++){
      final_sum += sum_gpu[i];
    }
    printf("The GPU sum is %d.\n", final_sum);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);
    printf("The GPU time is %ld.\n",used_time);
    int j;
    int sum_cpu = 0;
    clock_t start = clock();
    for (j=0; j < DATA_SIZE; j++){
      sum_cpu += data[j] * data[j];
    }
    clock_t end = clock()-start;
    printf("The CPU sum is %d.\n", sum_cpu);
    printf("The CPU time is %ld.\n",end);
    return 0;
  }
}
