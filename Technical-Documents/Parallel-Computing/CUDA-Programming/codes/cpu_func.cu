#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "gpu_func.h"
#include "cpu_func.h"

void read_B_data(hipfftComplex *C) {
    FILE * fp;

    if((fp = fopen("shujv.dat","rb")) == NULL){
        printf("file can not oepn!\n");
        exit(0);
    }

    unsigned char real[N];
    unsigned int line = 0;
    while((fread(real, sizeof(char), N, fp) > 0) & line < M * 2) { 
        unsigned char imag[N];
        fread(imag, sizeof(char), N, fp);

        int i;
        for(i = 0; i < N; i++) {
            C[(line/2) * N + i].x = real[i];
			C[(line/2) * N + i].y = imag[i];
			//printf("%f + %fi\n",C[(line/2) * N + i].x,C[(line/2) * N + i].y);
        }

        line += 2;
    }

    fclose(fp);
}

void read_S_data(hipfftComplex *C) {
    FILE * fp;

    if((fp = fopen("Weight_Final.txt","rb")) == NULL){
        printf("file can not oepn!\n");
        exit(0);
    }

    unsigned int line = 0;
    while(line < N) { 
        int i;
        for(i = 0; i < S; i++) {
            fscanf(fp,"%f",&C[line * S + i].x);
            fscanf(fp,"%f",&C[line * S + i].y);
			//printf("%f + %fi\n",C[line * S + i].x,C[line * S + i].y);
        }
        line++;
    }

    fclose(fp);
}

/* MatMultiply：CPU下矩阵乘法
*  a:第一个矩阵指针，表示a[m][n];
*  b:第二个矩阵指针，表示b[n][s];
*  result:结果矩阵，表示为result[m][s];
*/
void CPUMatMultiply(const hipfftComplex * a,const hipfftComplex * b, hipfftComplex *result,const int m,const int n,const int s) {
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < s; j++)
		{
			int index = i * s + j;
			result[index].x = 0;
			result[index].y = 0;

			//计算每一个元素的结果
			for (int k = 0; k < n; k++)
			{
				hipfftComplex temp;
				temp.x = a[i * n + k].x * b[k * s + j].x - a[i * n + k].y * b[k * s + j].y;
                temp.y = a[i * n + k].x * b[k * s + j].y + a[i * n + k].y * b[k * s + j].x;

				result[index].x += temp.x;
				result[index].y += temp.y;
            }
		}
	}
}

/* MatrTrans：CPU下矩阵转置
*  matrB: 转置完成的矩阵;
*  matrA: 需要转置的矩阵;
*/
void cpuMatrTrans(hipfftComplex *matrB, hipfftComplex *matrA, const int width, const int height) {
	int i, j;

    for (i = 0; i < height; i++)
        for (j = 0; j < width; j++) {
			matrB[j * height + i] = matrA[i * width + j];
		}
}

void compare_right(hipfftComplex *T, hipfftComplex *R) {
	int count = 0;
	float err = 0.00001;
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (abs(T[i * S + j].x - R[i * S + j].x) > err || abs(T[i * S + j].y - R[i * S + j].y) > err)
			{
				count++;
				printf("%d,%d:%f+%fi  %f+%fi\n",i,j,T[i * S + j].x,T[i * S + j].y,R[i * S + j].x,R[i * S + j].y);
				return;
			}
		}
	}

	printf("error count:%d\n",count);
}

void cpuDotMul(hipfftComplex *data, const int bi) {
	hipfftComplex *temp_data;
	temp_data = (hipfftComplex *)malloc (M * sizeof(hipfftComplex));
	memcpy(temp_data,data+bi*M,M*sizeof(hipfftComplex));
	hipfftComplex a,b;
	double real,imag;

	int i,j;
	for(i = 0; i < S; i++){
		for(j = 0; j < M; j++) {
			int offset = i*M + j;
			a.x = data[offset].x;
			a.y = data[offset].y;
			b.x = temp_data[offset%M].x;
			b.y = temp_data[offset%M].y;

            real = (double)a.x * (double)b.x - (double)a.y * (double)b.y;
			imag = (double)a.x * (double)b.y + (double)a.y * (double)b.x;

			data[offset].x = real;
			data[offset].y = imag;
		}
	}
}