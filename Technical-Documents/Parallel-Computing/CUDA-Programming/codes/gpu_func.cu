#include "hip/hip_runtime.h"
#include "gpu_func.h"

/* gpuMatMultWithTextureKernel：GPU下使用texture内存的矩阵乘法，并将结果存在转置后对应的位置
*  result：结果矩阵，表示为result[S][SM];
*  m：表示为矩阵A与矩阵result的行数
*  n：表示矩阵A的列数，矩阵B的行数
*  s：表示矩阵B和矩阵result的列数
*/
__global__ void gpuMatMultAndTransWithTextureKernel(hipfftComplex * result, const int m, const int n, const int s) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < m * s)
	{
		hipfftComplex a,b;
        hipfftComplex temp_result;
        temp_result.x = 0;
        temp_result.y = 0;
		for (int i = 0; i < n; i++)
		{
            a.x = tex1Dfetch(texA, y * n + i).x;
            a.y = tex1Dfetch(texA, y * n + i).y;
            b.x = tex1Dfetch(texB, i * s + x).x;
            b.y = tex1Dfetch(texB, i * s + x).y;

            hipfftComplex temp;
            temp.x = a.x * b.x - a.y * b.y;
            temp.y = a.x * b.y + a.y * b.x;

			temp_result.x += temp.x;
            temp_result.y += temp.y;
        }
		//result[offset] = temp_result;
		result[x * m + y] = temp_result;
	}
}

/* gpuDotMulWithTextureKernel：GPU下使用texture内存的矩阵点乘，并将结果存在转置后对应的位置
*  result：结果矩阵，表示为result[S][M];
*  m：表示为矩阵A与矩阵result的列数
*  s：表示矩阵B和矩阵result的行数
*/
__global__ void gpuDotMulWithTextureKernel(hipfftComplex * result, const int m, const int s) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	int bi_index = offset % m;
	hipfftComplex a,b;
	double real,imag;

	if (offset < m * s) {
		a.x = tex1Dfetch(texA, offset).x;
		a.y = tex1Dfetch(texA, offset).y;
		b.x = tex1Dfetch(texB, bi_index).x;
		b.y = tex1Dfetch(texB, bi_index).y;

        real = (double)a.x * (double)b.x - (double)a.y * (double)b.y;
		imag = (double)a.x * (double)b.y + (double)a.y * (double)b.x;
		
		result[offset].x = real;
		result[offset].y = imag;
	}
}

/* doAll: GPU下依次调用gpuMatMultAndTransWithTextureKernel和gpuDotMulWithTextureKernel
*         完成矩阵乘、转置、FFT、点乘操作
*  a: m*n 矩阵
*  b: n*s 矩阵
*  result: s*m 矩阵
*  bi: 与第bi行做点乘 
*/
hipError_t doAll(const hipfftComplex *a, const hipfftComplex *b, hipfftComplex *result, const int m, const int n, const int s, const int bi,hipfftHandle &plan_NX_Many) {
	hipfftComplex * dev_a;
	hipfftComplex * dev_b;
	hipfftComplex * dev_result;
	hipfftComplex * dev_bi_data;
	hipError_t cudaStatus;
	hipChannelFormatDesc desc = hipCreateChannelDesc<hipfftComplex>();

	//hipEvent_t gpuStart, gpuFinish;
	//float elapsedTime;
	//hipEventCreate(&gpuStart);
	//hipEventCreate(&gpuFinish);
	//hipEventRecord(gpuStart, 0);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, m * n * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, n * s * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, m * s * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_bi_data, m * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_bi_data failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, m * n * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, n * s * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, m * n * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, n * s * sizeof(hipfftComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	if ((m % BLOCK_SIZE != 0) && (s % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}
	
	//Mul%Trans
	gpuMatMultAndTransWithTextureKernel << <grid, block >> >(dev_result, m, n, s);

	//FFT
	checkCudaErrors(hipfftExecC2C(plan_NX_Many, dev_result, dev_result, HIPFFT_FORWARD));

	//DouMul
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	cudaStatus = hipBindTexture(NULL, texA, dev_result, desc, m * s * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_bi_data, desc, m * sizeof(hipfftComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bi_data, dev_result + bi * m, m * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	gpuDotMulWithTextureKernel << <grid, block >> >(dev_result, m, s);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, m * s * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

	//hipEventRecord(gpuFinish, 0);
	//hipEventSynchronize(gpuFinish);
	//hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	//printf("\nThe time of GPU do all is %f seconds.\n", elapsedTime / 1000.0);

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}

void test_FFT(hipfftComplex *data) {
    int i;
	for(i = 0;i<24;i++){
		hipfftComplex *d_fftData;
    	hipMalloc((void**)&d_fftData,M*sizeof(hipfftComplex));
    	hipMemcpy(d_fftData,data+i*M,M*sizeof(hipfftComplex),hipMemcpyHostToDevice);

		hipfftHandle plan;
		hipfftPlan1d(&plan,M,HIPFFT_C2C,1);
		hipfftExecC2C(plan,(hipfftComplex*)d_fftData,(hipfftComplex*)d_fftData,HIPFFT_FORWARD);
		hipDeviceSynchronize();
		hipMemcpy(data+i*M,d_fftData,M*sizeof(hipfftComplex),hipMemcpyDeviceToHost);
	}
}