#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "hip/device_functions.h"
#include ""
#include <chrono>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const int M = 16384;
const int N = 16;
const int S = 24;

//const char *file_C = "shujv.dat";
//const char *file_B = "temp_data.txt";
hipfftDoubleComplex *C;  // 16384 * 16
hipfftDoubleComplex *B;  // 16 * 24
hipfftDoubleComplex *X;  // 16384 * 24

/*
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
*/
/*
hipfftDoubleComplex hipCadd (hipfftDoubleComplex a, hipfftDoubleComplex b) {
    hipfftDoubleComplex temp;
    temp.x = a.x + b.x;
    temp.y = a.y + b.y;
    return temp;
}

hipfftDoubleComplex hipCmul (hipfftDoubleComplex a, hipfftDoubleComplex b) {
    hipFloatComplex temp;
    temp.x = a.x * b.x - a.y * b.y;
    temp.y = a.x * b.y + a.y * b.x;
    return temp;
}
*/

void CPUMatMultiply(const hipfftDoubleComplex * a,const hipfftDoubleComplex * b, hipfftDoubleComplex *result,const int M,const int N,const int S);
void cpuMatrTrans(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height);
void compare_right(hipfftDoubleComplex *T,hipfftDoubleComplex *R);
void test_FFT(hipfftDoubleComplex *data);
void cpuDotMul(hipfftDoubleComplex *data, const int bi);

// main function
void read_B_data(hipfftDoubleComplex *C);
void read_S_data(hipfftDoubleComplex *C);
__global__ void gpuMatMultAndTransWithSharedKernel(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S);
__global__ void gpuDotMulWithWithSharedKernel(const hipfftDoubleComplex *a, const hipfftDoubleComplex *bi, hipfftDoubleComplex *result, const int M, const int S);
//__global__ void gpuMatMultAndTransWithTextureKernel(int * result, const int M, const int N, const int S);
//__global__ void gpuDotMulWithTextureKernel(hipfftDoubleComplex * result, const int M, const int S);
hipError_t mulAndTransWithCudaShare(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S);
hipError_t dotMulWithSharedKernel(hipfftDoubleComplex *a, const int M, const int S, const int bi);
//hipError_t mulAndTransWithCudaTex(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S);
hipError_t gpuCuFFt(hipfftDoubleComplex *data, const int M, const int S);
//hipError_t DotMulWithTextureKernel(hipfftDoubleComplex *data, const int M, const int S, const int bi);

/*__global__ void notSoNaivaTransKernel(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height);
hipError_t matrMagicCuda(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height);*/

int main() {
    C = (hipfftDoubleComplex*)malloc(M*N*sizeof(hipfftDoubleComplex));
    B = (hipfftDoubleComplex*)malloc(N*S*sizeof(hipfftDoubleComplex));
	X = (hipfftDoubleComplex*)malloc(M*S*sizeof(hipfftDoubleComplex));
	// read data
    read_B_data(C);
	read_S_data(B);
	
	//gpu mul & trans
	//mulAndTransWithCudaTex(C,B,X,M,N,S); // X 24(S)*16384(M)
	mulAndTransWithCudaShare(C,B,X,M,N,S);

	// cpu mul & trans
	hipfftDoubleComplex *T;
	T = (hipfftDoubleComplex*)malloc(M*S*sizeof(hipfftDoubleComplex));
	CPUMatMultiply(C,B,T,M,N,S);
	hipfftDoubleComplex *trans_T;
	trans_T = (hipfftDoubleComplex*)malloc(M*S*sizeof(hipfftDoubleComplex));
	cpuMatrTrans(trans_T,T,S,M);
	compare_right(trans_T,X);

	//gpu fft
	gpuCuFFt(X,M,S);

	//test FFT
	test_FFT(trans_T);
	compare_right(trans_T,X);

	// DotMul
	//DotMulWithTextureKernel(X,M,S,0);
	/*printf("%lf+%lfi %lf+%lfi\n",X[209 * S + 0].x,X[209*S+0].y,trans_T[209*S+0].x,trans_T[209*S+0].y);
	printf("%lf+%lfi %lf+%lfi\n",X[209 * S % M].x,X[209 * S % M].y,trans_T[209 * S % M].x,trans_T[209 * S % M].y);
	printf("%lf+%lfi\n",X[209*S].x * X[209*S%M].x - X[209 * S].y * X[209*S%M].y, X[209*S].x * X[209*S%M].y + X[209 * S].y * X[209*S%M].x);*/
	dotMulWithSharedKernel(X,M,S,0);
	cpuDotMul(trans_T,0);
	//printf("%lf+%lfi %lf+%lfi\n",X[209 * S + 0].x,X[209*S+0].y,trans_T[209*S+0].x,trans_T[209*S+0].y);
	compare_right(trans_T,X);
}

void read_B_data(hipfftDoubleComplex *C) {
    FILE * fp;

    if((fp = fopen("shujv.dat","rb")) == NULL){
        printf("file can not oepn!\n");
        exit(0);
    }

    unsigned char real[N];
    unsigned int line = 0;
    while((fread(real, sizeof(char), N, fp) > 0) & line < M * 2) { 
        unsigned char imag[N];
        fread(imag, sizeof(char), N, fp);

        int i;
        for(i = 0; i < N; i++) {
            C[(line/2) * N + i].x = real[i];
			C[(line/2) * N + i].y = imag[i];
			//printf("%f + %fi\n",C[(line/2) * N + i].x,C[(line/2) * N + i].y);
        }

        line += 2;
    }

    fclose(fp);
}

void read_S_data(hipfftDoubleComplex *C) {
    FILE * fp;

    if((fp = fopen("Weight_Final.txt","rb")) == NULL){
        printf("file can not oepn!\n");
        exit(0);
    }

    unsigned int line = 0;
    while(line < N) { 
        int i;
        for(i = 0; i < S; i++) {
            fscanf(fp,"%lf",&C[line * S + i].x);
            fscanf(fp,"%lf",&C[line * S + i].y);
			//printf("%f + %fi\n",C[line * S + i].x,C[line * S + i].y);
        }
        line++;
    }

    fclose(fp);
}

/* gpuMatMultWithSharedKernel：GPU下使用shared内存的矩阵乘法
*  a:第一个矩阵指针，表示a[M][N]
*  b:第二个矩阵指针，表示b[N][S]
*  result:结果矩阵，表示result[S][M]
*/
template<int BLOCK_SIZE>
__global__ void gpuMatMultAndTransWithSharedKernel(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S) {
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;

	if ((thread_y + block_y * blockDim.y) * S + block_x * blockDim.x + thread_x >= M * S)
	{
		return;
	}

	const int begin_a = block_y * blockDim.y * N;
	const int end_a = begin_a + N - 1;
	const int step_a = blockDim.x;

	const int begin_b = block_x * blockDim.x;
	const int step_b = blockDim.y * S;

	hipfftDoubleComplex result_temp;
	result_temp.x = 0;
	result_temp.y = 0;

	int index_a,index_b;
	for (index_a = begin_a, index_b = begin_b;
		index_a < end_a; index_a += step_a, index_b += step_b)
	{
		__shared__ hipfftDoubleComplex SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ hipfftDoubleComplex SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		SubMat_A[thread_y][thread_x].x = a[index_a + thread_y * N + thread_x].x;
		SubMat_A[thread_y][thread_x].y = a[index_a + thread_y * N + thread_x].y;
		SubMat_B[thread_y][thread_x].x = b[index_b + thread_y * S + thread_x].x;
		SubMat_B[thread_y][thread_x].y = b[index_b + thread_y * S + thread_x].y;

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; i++)
		{	
			hipfftDoubleComplex temp;
			temp.x = SubMat_A[thread_y][i].x * SubMat_B[i][thread_x].x - SubMat_A[thread_y][i].y * SubMat_B[i][thread_x].y;
			temp.y = SubMat_A[thread_y][i].x * SubMat_B[i][thread_x].y + SubMat_A[thread_y][i].y * SubMat_B[i][thread_x].x;
			
			result_temp.x += temp.x;
			result_temp.y += temp.y;
		}

		__syncthreads();
	}

	int offset = (block_y * blockDim.y * S + begin_b) + thread_y * S + thread_x;
	int row = offset / S;
	int col = offset % S;
	result[col * M + row] = result_temp;
}

/* gpuDotMulWithWithSharedKernel：GPU下使用shared内存的矩阵点乘
*  a:目标矩阵指针，表示a[S][M]
*  bi:矩阵第i行指针，表示b[M]
*  result:结果矩阵，表示result[S][M]
*/
template<int BLOCK_SIZE>
__global__ void gpuDotMulWithWithSharedKernel(const hipfftDoubleComplex *a, const hipfftDoubleComplex *bi, hipfftDoubleComplex *result, const int M, const int S) {
	int block_x = blockIdx.x;
	int block_y = blockIdx.y;
	int thread_x = threadIdx.x;
	int thread_y = threadIdx.y;
	int offset = (thread_y + block_y * blockDim.y) * M + block_x * blockDim.x + thread_x;

	if (offset >= M * S)
	{
		return;
	}

	const int begin_a = block_y * blockDim.y * M;
	const int end_a = begin_a + M - 1;
	const int step_a = blockDim.x;

	int index_a;
	for (index_a = begin_a;
		index_a < end_a; index_a += step_a)
	{
		__shared__ hipfftDoubleComplex SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ hipfftDoubleComplex SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		SubMat_A[thread_y][thread_x].x = a[index_a + thread_y * M + thread_x].x;
		SubMat_A[thread_y][thread_x].y = a[index_a + thread_y * M + thread_x].y;
		SubMat_B[thread_y][thread_x].x = bi[(index_a + thread_y * M + thread_x) % M].x;
		SubMat_B[thread_y][thread_x].y = bi[(index_a + thread_y * M + thread_x) % M].y;

		__syncthreads();

		result[index_a + thread_y * M + thread_x].x = SubMat_A[thread_y][thread_x].x * SubMat_B[thread_y][thread_x].x - SubMat_A[thread_y][thread_x].y * SubMat_B[thread_y][thread_x].y;
		result[index_a + thread_y * M + thread_x].y = SubMat_A[thread_y][thread_x].x * SubMat_B[thread_y][thread_x].y + SubMat_A[thread_y][thread_x].y * SubMat_B[thread_y][thread_x].x;

		__syncthreads();
	}
}

/* gpuMatMultWithTextureKernel：GPU下使用texture内存的矩阵乘法，并将结果存在转置后对应的位置
*  result：结果矩阵，表示为result[M][S];
*  M：表示为矩阵A与矩阵result的行数
*  N：表示矩阵A的列数，矩阵B的行数
*  S：表示矩阵B和矩阵result的列数
*/
/*texture<hipfftDoubleComplex> texA;
texture<hipfftDoubleComplex> texB;
__global__ void gpuMatMultAndTransWithTextureKernel(hipfftDoubleComplex * result, const int M, const int N, const int S) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (offset < M * S)
	{
		hipfftDoubleComplex a,b;
        hipfftDoubleComplex temp_result;
        temp_result.x = 0;
        temp_result.y = 0;
		for (int i = 0; i < N; i++)
		{
            a.x = tex1Dfetch(texA, y * N + i).x;
            a.y = tex1Dfetch(texA, y * N + i).y;
            b.x = tex1Dfetch(texB, i * S + x).x;
            b.y = tex1Dfetch(texB, i * S + x).y;

            hipfftDoubleComplex temp;
            temp.x = a.x * b.x - a.y * b.y;
            temp.y = a.x * b.y + a.y * b.x;

			temp_result.x += temp.x;
            temp_result.y += temp.y;
        }
		//result[offset] = temp_result;
		result[x*M + y] = temp_result;
	}
}*/

hipError_t mulAndTransWithCudaShare(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S) {
	hipfftDoubleComplex *dev_a;
	hipfftDoubleComplex *dev_b;
	hipfftDoubleComplex *dev_result;
	const int BLOCK_SIZE = 8;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	gpuMatMultAndTransWithSharedKernel<8> << <grid, block >> >(dev_a, dev_b, dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}

//调用CUDA运行GPU矩阵乘法核函数
//将矩阵A与矩阵B绑定到纹理内存中
/*
hipError_t mulAndTransWithCudaTex(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex *result, const int M, const int N, const int S)
{
	hipfftDoubleComplex * dev_a;
	hipfftDoubleComplex * dev_b;
	hipfftDoubleComplex * dev_result;
    const int BLOCK_SIZE = 8;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(S / BLOCK_SIZE, M / BLOCK_SIZE);
	hipError_t cudaStatus;
	hipChannelFormatDesc desc = hipCreateChannelDesc<hipfftDoubleComplex>();

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_a, M * N * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_b, N * S * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, M * S * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texA, dev_a, desc, M * N * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_b, desc, N * S * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_a, a, M * N * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, N * S * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}

	gpuMatMultAndTransWithTextureKernel << <grid, block >> >(dev_result, M, N, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Mat Multiply & trans is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(result, dev_result, M * S * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_result);

	return cudaStatus;
}
*/

/* gpuCuFFt : GPU 下进行FFT
*  data:目标矩阵，S*M;
*/
hipError_t gpuCuFFt(hipfftDoubleComplex *data, const int M, const int S) {
	hipfftDoubleComplex *d_fftData;
	int number_M[1] = { M };
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&d_fftData, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc d_fftData failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_fftData, data, S * M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy d_fftData failed!\n");
		goto Error;
	}

	hipfftHandle plan_NX_Many;
	checkCudaErrors(hipfftCreate(&plan_NX_Many));
	checkCudaErrors(hipfftPlanMany(&plan_NX_Many, 1, number_M, NULL, 1, M, NULL, 1, M, HIPFFT_Z2Z, S));

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	checkCudaErrors(hipfftExecZ2Z(plan_NX_Many, d_fftData, d_fftData, HIPFFT_FORWARD));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cufft failed!\n");
		goto Error;
	}

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on FFT is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipMemcpy(data, d_fftData, S * M * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy result failed!\n");
		goto Error;
	}

Error:
	hipfftDestroy(plan_NX_Many);
	hipFree(d_fftData);

	return cudaStatus;
}

/*
__global__ void gpuDotMulWithTextureKernel(hipfftDoubleComplex * result, const int M, const int S) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	int bi_index = offset % M;
	hipfftDoubleComplex a,b;
	double real,imag;

	if (offset < M * S) {
		a.x = tex1Dfetch(texA, offset).x;
		a.y = tex1Dfetch(texA, offset).y;
		b.x = tex1Dfetch(texB, bi_index).x;
		b.y = tex1Dfetch(texB, bi_index).y;

        real = a.x * b.x - a.y * b.y;
		imag = a.x * b.y + a.y * b.x;
		
		result[offset].x = real;
		result[offset].y = imag;
	}
}

hipError_t DotMulWithTextureKernel(hipfftDoubleComplex *data, const int M, const int S, const int bi) {
	hipfftDoubleComplex * dev_data;
	hipfftDoubleComplex * dev_bi_data;
	hipfftDoubleComplex * dev_result;
    const int BLOCK_SIZE = 8;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(M / BLOCK_SIZE, S / BLOCK_SIZE);
	hipError_t cudaStatus;
	hipChannelFormatDesc desc = hipCreateChannelDesc<hipfftDoubleComplex>();

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_data, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_bi_data, M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texA, dev_data, desc, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texA failed!\n");
		goto Error;
	}

	cudaStatus = hipBindTexture(NULL, texB, dev_bi_data, desc, M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipBindTexture texB failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_data, data, S * M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bi_data, data + bi * M, M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	if ((M % BLOCK_SIZE != 0) && (S % BLOCK_SIZE != 0))
	{
		fprintf(stderr, "M or S can't be dividen by 16!\n");
		goto Error;
	}

	gpuDotMulWithTextureKernel << <grid, block >> >(dev_result, M, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Dot Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(data, dev_result, S * M * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipUnbindTexture(texA);
	hipUnbindTexture(texB);
	hipFree(dev_data);
	hipFree(dev_bi_data);
	hipFree(dev_result);

	return cudaStatus;
}
*/

hipError_t dotMulWithSharedKernel(hipfftDoubleComplex *a, const int M, const int S, const int bi) {
	hipfftDoubleComplex * dev_data;
	hipfftDoubleComplex * dev_bi_data;
	hipfftDoubleComplex * dev_result;
    const int BLOCK_SIZE = 8;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((S + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed! Do you have a CUDA_capable GPU installed?\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_data, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_bi_data, M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_b failed!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void **)&dev_result, S * M * sizeof(hipfftDoubleComplex));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc dev_result failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_data, a, S * M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "cudamemcpy dev_a failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_bi_data, a + bi * M, M * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy dev_b failed!\n");
		goto Error;
	}

	hipEvent_t gpuStart, gpuFinish;
	float elapsedTime;
	hipEventCreate(&gpuStart);
	hipEventCreate(&gpuFinish);
	hipEventRecord(gpuStart, 0);

	gpuDotMulWithWithSharedKernel<16> << <grid, block >> >(dev_data, dev_bi_data, dev_result, M, S);

	hipEventRecord(gpuFinish, 0);
	hipEventSynchronize(gpuFinish);
	hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
	printf("\nThe runing time of GPU on Dot Multiply is %f seconds.\n", elapsedTime / 1000.0);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "MulKernel launch failed: %s!\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize return Error code %d after Kernel launched!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(a, dev_result, S * M * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy result failed!\n");
		goto Error;
	}

Error:
	hipFree(dev_data);
	hipFree(dev_bi_data);
	hipFree(dev_result);

	return cudaStatus;
}

/* MatMultiply：CPU下矩阵乘法
*  a:第一个矩阵指针，表示a[M][N];
*  b:第二个矩阵指针，表示b[N][S];
*  result:结果矩阵，表示为result[M][S];
*/
void CPUMatMultiply(const hipfftDoubleComplex * a,const hipfftDoubleComplex * b, hipfftDoubleComplex *result,const int M,const int N,const int S) {
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			int index = i * S + j;
			result[index].x = 0;
			result[index].y = 0;

			//计算每一个元素的结果
			for (int k = 0; k < N; k++)
			{
				hipfftDoubleComplex temp;
				temp.x = a[i * N + k].x * b[k * S + j].x - a[i * N + k].y * b[k * S + j].y;
                temp.y = a[i * N + k].x * b[k * S + j].y + a[i * N + k].y * b[k * S + j].x;

				result[index].x += temp.x;
				result[index].y += temp.y;
            }
		}
	}
}

/* MatrTrans：CPU下矩阵转置
*  matrB: 转置完成的矩阵;
*  matrA: 需要转置的矩阵;
*/
void cpuMatrTrans(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height) {
	int i, j;

    for (i = 0; i < height; i++)
        for (j = 0; j < width; j++) {
			matrB[j * height + i] = matrA[i * width + j];
		}
}

void compare_right(hipfftDoubleComplex *T,hipfftDoubleComplex *R) {
	int count = 0;
	float err = 0.00001;
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < S; j++)
		{
			if (abs(T[i * S + j].x - R[i * S + j].x) > err || abs(T[i * S + j].y - R[i * S + j].y) > err)
			{
				count++;
				//printf("%d,%d:%f+%fi  %f+%fi\n",i,j,T[i * S + j].x,T[i * S + j].y,R[i * S + j].x,R[i * S + j].y);
			}
		}
	}

	printf("error count:%d\n",count);
}

void test_FFT(hipfftDoubleComplex *data) {
    int i;
	for(i = 0; i < S; i++){
		hipfftDoubleComplex *d_fftData;
    	hipMalloc((void**)&d_fftData,M*sizeof(hipfftDoubleComplex));
    	hipMemcpy(d_fftData,data+i*M,M*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice);

		hipfftHandle plan;
		hipfftPlan1d(&plan,M,HIPFFT_Z2Z,1);
		hipfftExecZ2Z(plan,(hipfftDoubleComplex*)d_fftData,(hipfftDoubleComplex*)d_fftData,HIPFFT_FORWARD);
		hipDeviceSynchronize();
		hipMemcpy(data+i*M,d_fftData,M*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
	}
}

void cpuDotMul(hipfftDoubleComplex *data, const int bi) {
	hipfftDoubleComplex *temp_data;
	temp_data = (hipfftDoubleComplex *)malloc (M * sizeof(hipfftDoubleComplex));
	memcpy(temp_data,data+bi*M,M*sizeof(hipfftDoubleComplex));
	hipfftDoubleComplex a,b;
	double real,imag;

	int i,j;
	for(i = 0; i < S; i++){
		for(j = 0; j < M; j++) {
			int offset = i * M + j;
			a.x = data[offset].x;
			a.y = data[offset].y;
			b.x = temp_data[offset%M].x;
			b.y = temp_data[offset%M].y;

            real = a.x * b.x - a.y * b.y;
			imag = a.x * b.y + a.y * b.x;

			data[offset].x = real;
			data[offset].y = imag;
		}
	}
}

/*
__global__ void notSoNaivaTransKernel(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height)
{
    __shared__ hipfftDoubleComplex tile[TILE_DIM][TILE_DIM + 1];
    int ciIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int riIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int coIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    int roIndex = blockIdx.x * TILE_DIM + threadIdx.y;
    int index_in = ciIndex + (riIndex)* width;
    int index_out = coIndex + (roIndex)* height;

    int i;
    for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((ciIndex<width) && (riIndex+i < height)) {
			tile[threadIdx.y + i][threadIdx.x] = matrA[index_in + i * width];
		}
    __syncthreads();

    for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        if ((coIndex<height) && (roIndex+i < width)) {
			matrB[index_out + i*height] = tile[threadIdx.x][threadIdx.y + i];
		}
    __syncthreads();
}*/

/*
hipError_t matrMagicCuda(hipfftDoubleComplex *matrB, hipfftDoubleComplex *matrA, const int width, const int height)
{
    float elapsed = 0;
    hipfftDoubleComplex *dev_matrA;
    hipfftDoubleComplex *dev_matrB;
    hipError_t cudaStatus;
    dim3 dim_grid, dim_block;

    int size = width * height;

    dim_block.x = TILE_DIM;
    dim_block.y = BLOCK_ROWS;
    dim_block.z = 1;

    dim_grid.x = (width + TILE_DIM - 1) / TILE_DIM;
    dim_grid.y = (height + TILE_DIM - 1) / TILE_DIM;
    dim_grid.z = 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three matrix
    cudaStatus = hipMalloc((void**)&dev_matrA, size * sizeof(hipfftDoubleComplex));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_matrB, size * sizeof(hipfftDoubleComplex));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input matrix from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_matrA, matrA, size * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	hipEventRecord(start);
	// Launch a kernel on the GPU with one thread for each element.
	notSoNaivaTransKernel << <dim_grid, dim_block >> >(dev_matrB, dev_matrA, width, height);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsed, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("GPU Trans with SM Time: %f s\n", elapsed  / 1000.0);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output matrix from GPU buffer to host memory.
    cudaStatus = hipMemcpy(matrB, dev_matrB, size * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_matrB);
    hipFree(dev_matrA);

    return cudaStatus;
}
*/
