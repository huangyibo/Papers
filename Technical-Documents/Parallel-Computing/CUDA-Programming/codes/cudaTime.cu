#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
  int driverVersion;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  hipDriverGetVersion(&driverVersion);//CUDA代码
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float elapsedTime;   //存放时间间隔
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("The time elapsed time is %5.5f\n", elapsedTime);
  printf("CUDA Driver Version:%d.%d\n",driverVersion/1000,driverVersion/100);
  return 0;
}
