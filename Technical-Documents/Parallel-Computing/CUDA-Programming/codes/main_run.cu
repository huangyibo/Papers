#include "hip/hip_runtime.h"
#include "stdio.h"
#include <time.h>
#include <math.h>
#include "gpu_func.cu"
#include "cpu_func.cu"

int main() {

    C = (hipfftComplex*)malloc(M*N*sizeof(hipfftComplex));
    B = (hipfftComplex*)malloc(N*S*sizeof(hipfftComplex));
	X = (hipfftComplex*)malloc(M*S*sizeof(hipfftComplex));

	hipfftHandle plan_NX_Many;
	int number_M[1] = { M };
	checkCudaErrors(hipfftCreate(&plan_NX_Many));
	checkCudaErrors(hipfftPlanMany(&plan_NX_Many, 1, number_M, NULL, 1, M, NULL, 1, M, HIPFFT_C2C, S));
	
	/*
	// read data
    read_B_data(C);
	read_S_data(B);
	
	//gpu mul & trans
	mulAndTransWithCudaTex(C,B,X,M,N,S); // X 24(S)*16384(M)

	// cpu mul & trans
	hipfftComplex *T;
	T = (hipfftComplex*)malloc(M*S*sizeof(hipfftComplex));
	CPUMatMultiply(C,B,T,M,N,S);
	hipfftComplex *trans_T;
	trans_T = (hipfftComplex*)malloc(M*S*sizeof(hipfftComplex));
	cpuMatrTrans(trans_T,T,S,M);
	compare_right(trans_T,X);

	//gpu fft
	gpuCuFFt(X,M,S);

	//test FFT
	test_FFT(trans_T);
	compare_right(trans_T,X);

	// DotMul
	DotMulWithTextureKernel(X,M,S,0);
	cpuDotMul(trans_T,0);
	compare_right(trans_T,X);
	*/

	clock_t start,finish;
	/*
	start = clock();
	read_B_data(C);
	read_S_data(B);
	mulAndTransWithCudaTex(C,B,X,M,N,S); // X 24(S)*16384(M)
	gpuCuFFt(X,M,S);
	DotMulWithTextureKernel(X,M,S,0);
	finish = clock();
	printf("No1: total time is %lf s\n",(double)(finish-start)/CLOCKS_PER_SEC); 
	*/

	int count = 0;
	start = clock();
	//read_B_data(C);
	read_S_data(B);
	//doAll(C,B,X,M,N,S,0,plan_NX_Many);
	//finish = clock();
	//printf("total time is %lf s\n",(double)(finish-start)/CLOCKS_PER_SEC);
	while(((double)((finish = clock()) - start)/CLOCKS_PER_SEC) < 1.0) {
		read_B_data(C);
		doAll(C,B,X,M,N,S,0,plan_NX_Many);
		count++;
	}
	double total_time = (double)(finish - start) / CLOCKS_PER_SEC;
	int total_size = (count*(M*N*sizeof(float)*2) + (N*S*sizeof(float)*2)) / 1024 / 1024;
	printf("run time\tMat count\tdata size\tthoughput\n");
	printf("%lfs\t%d\t\t%dMb\t\t%lfMb/s\n",total_time,count,total_size,(double)total_size/total_time);

	hipfftComplex *T;
	T = (hipfftComplex*)malloc(M*S*sizeof(hipfftComplex));
	CPUMatMultiply(C,B,T,M,N,S);
	hipfftComplex *trans_T;
	trans_T = (hipfftComplex*)malloc(M*S*sizeof(hipfftComplex));
	cpuMatrTrans(trans_T,T,S,M);
	test_FFT(trans_T);
	cpuDotMul(trans_T,0);
	compare_right(trans_T,X);

	free(C);free(B);free(X);
	//free(T);free(trans_T);
}
