#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdbool.h> 
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}
 
bool InitCUDA()
{
    //used to count the device numbers
    int count;
 
    // get the cuda device count
    hipGetDeviceCount(&count);
    printf("cuda device count: %d\n",count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return 1;
    }
 
    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.totalGlobalMem >= 1000) {
                printDeviceProp(prop);
                hipSetDevice(i);
                return 0;
            }
        }
    }
 
    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return 1;
    }
 
    return 0;
}
 
int main(int argc, char const *argv[])
{
    if (InitCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}
