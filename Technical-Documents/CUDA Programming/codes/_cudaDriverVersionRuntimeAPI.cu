#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
  int driverVersion;
  hipDriverGetVersion(&driverVersion);
  printf("CUDA Driver Version:%d.%d\n",driverVersion/1000,driverVersion/100);
  return 0;
}
