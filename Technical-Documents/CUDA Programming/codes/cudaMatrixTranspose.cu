#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include <chrono>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8
#define BLOCK_COLS 32

hipError_t matrMagicCuda(float *matrB, float *matrA, const int width, const int height, const int nreps, const int operation);
void cpuMatrTrans(float *matrB, float *matrA, const int width, const int height, const int nreps);
__global__ void naiveTransKernel(float *matrB, float *matrA, const int width, const int height, const int nreps);
__global__ void notSoNaivaTransKernel(float *matrB, float *matrA, const int width, const int height, const int nreps);

int main()
{
    int i, width, height, nreps, size, wrong, correct;
    double cpuTime, cpuBandwidth;
    hipError_t cudaStatus;

    float *matrA, *matrATC, *matrATG, *matrAC;

    srand(time(NULL));

    nreps = 10000;
    width = 500;
    height = 100;


    size = width * height;

    matrA = (float*)malloc(size * sizeof(float)); // matrix A
    matrAC = (float*)malloc(size * sizeof(float)); // matrix A copied
    matrATC = (float*)malloc(size * sizeof(float)); // matrix A transposed by CPU
    matrATG = (float*)malloc(size * sizeof(float)); // matrix A transposed by GPU

    for (i = 0; i < size; i++)
    {
        matrA[i] = (float)i;
    }

    auto start = std::chrono::high_resolution_clock::now();

    //CPU Transpose
    cpuMatrTrans(matrATC, matrA, width, height, nreps);

    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> diff = end - start;
    cpuTime = (diff.count() * 1000) / nreps;
    cpuBandwidth = (sizeof(float) * size * 2) / (cpuTime * 1000000);//scaling from ms to s and B to GB doen implicitly, shortened in fraction, times two for read and write
    printf("Avg. CPU Transpose Time: %f ms, Bandwidth: %f GB/s\n\n", cpuTime, cpuBandwidth);

    correct = 0;
    wrong = 0;

    //Naive transpose
    memset(matrATG, 0, size*sizeof(float));
    matrMagicCuda(matrATG, matrA, width, height, nreps, 1);

    //Check if calc was correct
    for (i = 0; i < size; i++)
    {
        if (matrATC[i] != matrATG[i])
        {
            /*printf("ERROR - %d - ATC:%f - ATG:%f\n\n", i, matrATC[i], matrATG[i]);
            return;*/
            wrong++;
        }
        else
        {
            correct++;
        }
    }

    printf("\tCorrect: %d, Wrong: %d\n\n", correct, wrong);
    correct = 0;
    wrong = 0;

    //Transpose with shared memory
    memset(matrATG, 0, size*sizeof(float));
    matrMagicCuda(matrATG, matrA, width, height, nreps, 2);

    //Check if calc was correct
    for (i = 0; i < size; i++)
    {
        if (matrATC[i] != matrATG[i])
        {
            /*printf("ERROR - %d - ATC:%f - ATG:%f\n\n", i, matrATC[i], matrATG[i]);
            return;*/
            wrong++;
        }
        else
        {
            correct++;
        }
    }

    //printf("\tTranspose with SM on GPU was executed correctly.\n\n");
    printf("\tCorrect: %d, Wrong: %d\n\n", correct, wrong);
    correct = 0;
    wrong = 0;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!\n");
        return 1;
    }

    return 0;
}

hipError_t matrMagicCuda(float *matrB, float *matrA, const int width, const int height, const int nreps, const int operation)
{
    float elapsed = 0;
    float *dev_matrA = 0;
    float *dev_matrB = 0;
    hipError_t cudaStatus;
    dim3 dim_grid, dim_block;
    double gpuBandwidth;

    int size = width * height;

    dim_block.x = TILE_DIM;
    dim_block.y = BLOCK_ROWS;
    dim_block.z = 1;

    dim_grid.x = (width + TILE_DIM - 1) / TILE_DIM;
    dim_grid.y = (height + TILE_DIM - 1) / TILE_DIM;
    dim_grid.z = 1;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three matrix
    cudaStatus = hipMalloc((void**)&dev_matrA, size * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_matrB, size * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input matrix from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_matrA, matrA, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    hipMemset(dev_matrB, 0, size * sizeof(float));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    switch (operation)
    {
        case(1):
        {
            hipEventRecord(start);
            // Launch a kernel on the GPU with one thread for each element.
            naiveTransKernel << <dim_grid, dim_block >> >(dev_matrB, dev_matrA, width, height, nreps);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            hipEventElapsedTime(&elapsed, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);

            elapsed /= nreps;

            gpuBandwidth = (sizeof(float) * size * 2) / (elapsed * 1000000);//scaling from ms to s and B to GB doen implicitly, shortened in fraction, times two for read and write
            printf("Avg. GPU Naive Trans Time: %f ms, bandwidth: %f GB/s\n", elapsed, gpuBandwidth);

            break;
        }

        case(2):
        {
            hipEventRecord(start);
            // Launch a kernel on the GPU with one thread for each element.
            notSoNaivaTransKernel << <dim_grid, dim_block >> >(dev_matrB, dev_matrA, width, height, nreps);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            hipEventElapsedTime(&elapsed, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);

            elapsed /= nreps;

            gpuBandwidth = (sizeof(float) * size * 2) / (elapsed * 1000000);//scaling from ms to s and B to GB doen implicitly, shortened in fraction, times two for read and write
            printf("Avg. GPU Trans with SM Time: %f ms, bandwidth: %f GB/s\n", elapsed, gpuBandwidth);

            break;
        }

    default:
        printf("No matching opcode was found.\n");
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output matrix from GPU buffer to host memory.
    cudaStatus = hipMemcpy(matrB, dev_matrB, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_matrB);
    hipFree(dev_matrA);

    return cudaStatus;
}

void cpuMatrTrans(float *matrB, float *matrA, const int width, const int height, const int nreps)
{
    int i, j, r;

#pragma unroll
    for (r = 0; r < nreps; r++)
#pragma unroll
        for (i = 0; i < height; i++)
#pragma unroll
            for (j = 0; j < width; j++)
                matrB[j * height + i] = matrA[i * width + j];
}

__global__ void naiveTransKernel(float *matrB, float *matrA, const int width, const int height, const int nreps)
{
    int i, r;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int index_in = col + width * row;
    int index_out = row + height * col;

#pragma unroll
    for (r = 0; r < nreps; r++)
#pragma unroll
        for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            if ((row+i<height) && (col < width))
                matrB[index_out + i] = matrA[index_in + i * width];
}

__global__ void notSoNaivaTransKernel(float *matrB, float *matrA, const int width, const int height, const int nreps)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];
    int ciIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int riIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int coIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    int roIndex = blockIdx.x * TILE_DIM + threadIdx.y;
    int index_in = ciIndex + (riIndex)* width;
    int index_out = coIndex + (roIndex)* height;


    int r, i;
#pragma unroll
    for (r = 0; r < nreps; r++)
    {
#pragma unroll
        for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            if ((ciIndex<width) && (riIndex+i < height))
              tile[threadIdx.y + i][threadIdx.x] = matrA[index_in + i * width];
        __syncthreads();

#pragma unroll
        for (i = 0; i < TILE_DIM; i += BLOCK_ROWS)
            if ((coIndex<height) && (roIndex+i < width))
               matrB[index_out + i*height] = tile[threadIdx.x][threadIdx.y + i];
        __syncthreads();
    }
}
