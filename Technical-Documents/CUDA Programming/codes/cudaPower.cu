#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define DATA_SIZE 1048576     //该数值随便指定
int data[DATA_SIZE];    //全局data数组，要计算数组中数据的平方和

/*
  * @author Junpeng Zhu
  * @功能  为当前程序设置一个合适的GPU，当然还有其它的方式能够实现该功能，比如设置环境变量
*/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);   //统计计算机中支持CUDA的显卡的个数
  if (count == 0){
    fprintf(stderr, "There is no device!\n");
    return false;
  }
  
  int i;
  for (i = 0; i < count; i++){
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop,i) == hipSuccess){
      if (prop.major >= 1 and prop.minor >= 0){
        break;
      }
    }
  }
  if (i == count){
    fprintf(stderr,"There is no device support CUDA 1.x.\n");
    return false;
  }

  hipSetDevice(i);   //设定满足>=1.0的计算设备为当前程序需要的计算设备
  return true;
}

/*
  * @author Junpeng Zhu
  * @功能  随机数生成函数，生成0-9的以内的整数
*/
void GenerateNumbers(int *number, int size){
  for (int i =0; i < size; i++){
    number[i] = rand() % 10;
  }
}


int main(){
  if (!InitCUDA()){
    return 0;
  }else{
    printf("CUDA initialized.\n");
    GenerateNumbers(data, DATA_SIZE);   //生成指定规模的随机数据，所有数据范围均在0-9
    int *gpudata, *result;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE); //在device上分配一块linear地址空间，地址为&gpudata，每个单元大小为int，总共DATA_SIZE个单元
    hipMalloc((void **) &result, sizeof(int));  //在device上分配内存，大小为int
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);   //从host的data中将DATA_SIZE个单元的数据复制到device的gpudata中，这是输入数据，需要从host中获取
    
    
    return 0;
  }
}
