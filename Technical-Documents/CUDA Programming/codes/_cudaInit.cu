#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdbool.h> 
#include <string.h>

//打印GPU信息
void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %ld.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %ld.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %ld.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %ld.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %ld.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}


bool initCUDA()
{
    //used to count the device numbers
    int count;
 
    // get the cuda device count
    hipGetDeviceCount(&count);
    printf("cuda device count: %d\n",count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    // 寻找指定的Tesla GPU
    int i;
    for (i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            printf("Device Name : %s.\n", prop.name);
            if ( (strcasecmp(prop.name, "Tesla P40")) == 0) {
               printf("Device Name : %s.\n", prop.name);  //被选定的设备在打印时，应该被连续打印两次
               break;
            }
        }
    }
 
    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}
 
int main(int argc, char const *argv[])
{
    if (initCUDA()) {
        printf("CUDA initialized.\n");
    }
 
    return 0;
}
